
#include <hip/hip_runtime.h>
# include <iostream>
# include <cstdlib>

using namespace std;

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

int main(int argc, const char * const * argv)
{
	int a, b, c, *dev_c;

	if (argc != 3) {
		cout << "please input two numbers a and b." << endl;
		return 1;
	}

	a = atoi(argv[1]);
	b = atoi(argv[2]);

	hipMalloc((void**)&dev_c, sizeof(int));

	add<<<1,1>>>(a, b, dev_c);

	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_c);

	cout << a << "+" << b << "=" << c <<"." << endl;

	return 0;
}
